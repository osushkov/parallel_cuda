
#include "Util.hpp"
#include <iostream>
#include <cassert>

using namespace cuda;

void util::OutputError(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    std::cerr << "GPU error: " << hipGetErrorString(code) << " "
        << file << "(" << line << ")" << std::endl;
    exit(code);
  }
}

void *util::AllocPinned(size_t bufSize) {
  void* result = nullptr;

  hipError_t err = hipHostAlloc(&result, bufSize, hipHostMallocPortable);
  CheckError(err);
  assert(result != nullptr);

  return result;
}

void util::FreePinned(void *buf) {
  assert(buf != nullptr);
  hipError_t err = hipHostFree(buf);
  CheckError(err);
}
