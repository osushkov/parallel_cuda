
#include "Util.hpp"
#include "Util.cuh"
#include <iostream>
#include <cassert>
#include <hip/hip_runtime.h>

using namespace cuda;

void util::OutputError(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    std::cerr << "GPU error: " << hipGetErrorString(code) << " "
        << file << "(" << line << ")" << std::endl;
    exit(code);
  }
}

void *util::AllocPinned(size_t bufSize) {
  void* result = nullptr;

  hipError_t err = hipHostAlloc(&result, bufSize, hipHostMallocPortable);
  CheckError(err);
  assert(result != nullptr);

  return result;
}

void util::FreePinned(void *buf) {
  assert(buf != nullptr);
  hipError_t err = hipHostFree(buf);
  CheckError(err);
}
