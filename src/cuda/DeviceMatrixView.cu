
#include "DeviceMatrixView.hpp"
#include "Util.cuh"
#include <cassert>
#include <hip/hip_runtime.h>
#include <iostream>

DeviceMatrixView DeviceMatrixView::New(int rows, int cols) {
  assert(rows > 0 && cols > 0);

  DeviceMatrixView result;
  result.rows = rows;
  result.cols = cols;

  size_t width = cols * sizeof(float);
  size_t height = rows;

  hipError_t err = hipMallocPitch(&result.data, &result.pitch, width, height);
  CheckError(err);

  return result;
}

void DeviceMatrixView::Delete(DeviceMatrixView &dmv) {
  hipError_t err = hipFree(dmv.data);
  CheckError(err);
  dmv.data = nullptr;
}
