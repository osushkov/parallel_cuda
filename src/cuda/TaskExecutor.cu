
#include "TaskExecutor.hpp"
#include "Util.cuh"
#include "CudaKernel.hpp"
#include <hip/hip_runtime.h>
#include <utility>
#include <iostream>

using namespace cuda;
using namespace std;

struct TaskExecutor::TaskExecutorImpl {
  StreamId nextStreamId;
  vector<pair<StreamId, hipStream_t>> streamsMap;

  TaskExecutorImpl() : nextStreamId(1) {};

  ~TaskExecutorImpl() {
    for (auto &s : streamsMap) {
      hipStreamDestroy(s.second);
    }
  }

  StreamId CreateStream(void) {
    hipStream_t newStream;
    hipStreamCreateWithFlags(&newStream, hipStreamNonBlocking);
    streamsMap.emplace_back(nextStreamId++, newStream);
    return streamsMap.back().first;
  }

  void ExecuteTask(Task &task, StreamId stream) {
    invokeTask(task, stream);
    syncStream(stream);
  }

  void ExecuteTasks(vector<Task> &tasks, StreamId stream){
    for (auto &t : tasks) {
      invokeTask(t, stream);
    }
    syncStream(stream);
  }

  void invokeTask(Task &task, StreamId stream) {
    if (task.type == TaskType::DH_COPY) {
      hipError_t err = hipMemcpy2DAsync(
        task.dhCopy.dst.data, task.dhCopy.dst.cols * sizeof(float),
        task.dhCopy.src.data, task.dhCopy.src.pitch,
        task.dhCopy.src.cols * sizeof(float), task.dhCopy.src.rows,
        hipMemcpyDeviceToHost, getStream(stream));

      CheckError(err);
    } else if (task.type == TaskType::HD_COPY) {
      hipError_t err = hipMemcpy2DAsync(
        task.hdCopy.dst.data, task.hdCopy.dst.pitch,
        task.hdCopy.src.data, task.hdCopy.src.cols * sizeof(float),
        task.hdCopy.src.cols * sizeof(float), task.hdCopy.src.rows,
        hipMemcpyHostToDevice, getStream(stream));

      CheckError(err);

    } else if (task.type == TaskType::MULT) {
      CudaKernel::Multiply(task.mult.lhs, task.mult.rhs, task.mult.out, getStream(stream));
    } else {
      assert(false);
    }
  }

  void syncStream(StreamId stream) {
    hipStreamSynchronize(getStream(stream));
  }

  hipStream_t getStream(StreamId id) {
    assert(!streamsMap.empty());
    for (const auto &e : streamsMap) {
      if (e.first == id) {
        return e.second;
      }
    }
    assert(false);
    return 0;
  }
};

TaskExecutor::TaskExecutor() : impl(new TaskExecutorImpl()) {}
TaskExecutor::~TaskExecutor() = default;

StreamId TaskExecutor::CreateStream(void) { return impl->CreateStream(); }

void TaskExecutor::ExecuteTask(Task &task, StreamId stream) {
  impl->ExecuteTask(task, stream);
}

void TaskExecutor::ExecuteTasks(vector<Task> &tasks, StreamId stream) {
  impl->ExecuteTasks(tasks, stream);
}
