
#include "TaskExecutor.hpp"
#include <hip/hip_runtime.h>
#include <utility>

using namespace cuda;
using namespace std;

struct TaskExecutor::TaskExecutorImpl {
  StreamId nextStreamId;
  vector<pair<StreamId, hipStream_t>> streamsMap;

  TaskExecutorImpl() : nextStreamId(1) {};

  ~TaskExecutorImpl() {
    for (auto &s : streamsMap) {
      hipStreamDestroy(s.second);
    }
  }

  StreamId CreateStream(void) {
    hipStream_t newStream;
    hipStreamCreate(&newStream);
    streamsMap.emplace_back(nextStreamId++, newStream);
    return streamsMap.back().first;
  }

  void ExecuteTask(const Task &task, StreamId stream) {
    invokeTask(task, stream);
    syncStream(stream);
  }

  void ExecuteTasks(const vector<Task> &tasks, StreamId stream){
    for (const auto &t : tasks) {
      invokeTask(t, stream);
    }
    syncStream(stream);
  }

  void invokeTask(const Task &task, StreamId stream) {

  }

  void syncStream(StreamId stream) {
    hipStreamSynchronize(getStream(stream));
  }

  hipStream_t getStream(StreamId id) {
    assert(!streamsMap.empty());
    for (const auto &e : streamsMap) {
      if (e.first == id) {
        return e.second;
      }
    }
    assert(false);
    return 0;
  }
};

TaskExecutor::TaskExecutor() : impl(new TaskExecutorImpl()) {}
TaskExecutor::~TaskExecutor() = default;

StreamId TaskExecutor::CreateStream(void) { return impl->CreateStream(); }

void TaskExecutor::ExecuteTask(const Task &task, StreamId stream) {
  impl->ExecuteTask(task, stream);
}

void TaskExecutor::ExecuteTasks(const vector<Task> &tasks, StreamId stream) {
  impl->ExecuteTasks(tasks, stream);
}
